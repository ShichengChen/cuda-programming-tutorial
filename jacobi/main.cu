#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include "definitions.cuh"
#include <time.h>
#include "stdio.h"

//Number of elements on which to perform CFD
unsigned int Ni = 512; // Y elements
unsigned int Nj = 512; // X elements
unsigned int nIterations = 10000; // No Of Iterations
unsigned int kernelVersion =2; // Decides which GPU kernel version to call (Set it to 1 or 2)

int main(int argc, char** argv)
{

    //Variables for Timing
    float cpuTime, gpuTime;

    // CPU and GPU Pointers ( d_XX : refers to pointer pointing to GPU memory. This is just a convention)
    float *t = NULL, *t_prev = NULL;
    float *d_t = NULL,*d_t_prev= NULL;

    parseCommandLineArguments(argc, (char **)argv);
    printf("\n Ni= %d, Nj=%d nIteration=%d",Ni,Nj,nIterations);

    unsigned int size = Ni * Nj * sizeof(float);

    if(!initializeCPU(&t, &t_prev) )
    {
        printf("\n Error in allocating memory on CPU!!!");
        unInitializeCPU(&t, &t_prev);
        getchar();
        return 0;
    }

    if (!initializeGPU(&d_t, &d_t_prev))
    {
        printf("\n Error in allocating memory on GPU!!!");
        unInitializeCPU(&t, &t_prev);
        unInitializeGPU(&d_t, &d_t_prev);
        return 0;
    }

    //Perform CFD on CPU
    performCPUCFD(t,t_prev, &cpuTime);

    // To temporarily store CPU data. This is just for comparing with GPU output
    float *tempBuffer = (float*) calloc(Ni*Nj, sizeof(float));
    memcpy(tempBuffer, t_prev, size);

    //Perform CFD on GPU
    if(!performGPUCFD(d_t,d_t_prev, t, t_prev, &gpuTime))
    {
        printf("\n GPU Kernel failed !!!");
        unInitializeCPU(&t, &t_prev);
        unInitializeGPU(&d_t, &d_t_prev);
        if(tempBuffer !=NULL)
            free(tempBuffer);
        return 0;
    }

    printf("\n Is host equal to device = %d", checkHostEqualsDevice(tempBuffer,t));
    printf("\n Speedup = %fx", (float)(cpuTime/gpuTime));

    unInitializeCPU(&t, &t_prev);
    unInitializeGPU(&d_t, &d_t_prev);

    if(tempBuffer !=NULL)
        free(tempBuffer);

    printf("\n Finished Processing!!!");
    getchar();

}


void parseCommandLineArguments(int argc, char**argv)
{
    if (argc >= 1)
    {
        for (int i=1; i < argc; i++)
        {
            int bFirstArgIsParam = false;
            int string_start = 0;
            while (argv[i][string_start] == '-')
                string_start++;
            char *string_argv = &argv[i][string_start];

            if (!STRNCASECMP(string_argv, "Ni=", 3))
            {
                bFirstArgIsParam = true;
                Ni = atoi(&string_argv[3]);
                continue;
            }
            if (!STRNCASECMP(string_argv, "Nj=", 3))
            {
                bFirstArgIsParam = true;
                Nj = atoi(&string_argv[3]);
                continue;
            }
            if (!STRNCASECMP(string_argv, "iterations=", 11))
            {
                bFirstArgIsParam = true;
                nIterations = atoi(&string_argv[11]);
                continue;
            }
            if (!STRNCASECMP(string_argv, "kernel=", 7))
            {
                bFirstArgIsParam = true;
                kernelVersion = atoi(&string_argv[7]);
                continue;
            }

            if (!bFirstArgIsParam)
            {
                printf("Invalid arguments\n");
                for (int n=0; n < argc; n++)
                {
                    printf("argv[%d] = %s\n", n, argv[n]);
                }
                printf("\n");
                exit(0);
            }
        }
    }

    if(( Ni % THREADS_PER_BLOCK_Y != 0)  || (Nj % THREADS_PER_BLOCK_X != 0))
    {
        fprintf(stderr, "Please specify Ni & Nj as multiple of 16 !!!!");
        getchar();
        exit(0);
    }
}

int initializeCPU(float **t, float **t_prev)
{
    *t = (float*) calloc(Ni*Nj, sizeof(float));
    *t_prev = (float*) calloc(Ni*Nj, sizeof(float));

    if((*t)==NULL || (*t_prev) == NULL)
        return 0;
    else
        return 1;
}

void unInitializeCPU(float **t, float **t_prev)
{
    if((*t) !=NULL)
        free(*t);
    if((*t_prev) != NULL)
        free(*t_prev);
}

int initializeGPU(float **d_t, float **d_t_prev)
{

    unsigned int size = Ni * Nj * sizeof(float);



    // Choose which GPU to run on, change this on a multi-GPU system.
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        getchar();
        return 0;
    }
    // Allocate GPU buffers.
    cudaStatus = hipMalloc((void**)&(*d_t), size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        getchar();
        return 0;
    }

    // Allocate GPU buffers   .
    cudaStatus = hipMalloc((void**)&(*d_t_prev), size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        getchar();
        return 0;
    }

    // Memset GPU buffers
    cudaStatus = hipMemset((*d_t),0, size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemset failed!");
        getchar();
        return 0;
    }

    // Memset GPU buffers
    cudaStatus = hipMemset((*d_t_prev),0, size);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemset failed!");
        getchar();
        return 0;
    }

    return 1;
}


void unInitializeGPU(float **d_t, float **d_t_prev)
{
    hipError_t cudaStatus;

    if((*d_t)!=NULL)
        cudaStatus = hipFree((*d_t));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipFree failed!");
        return;
    }

    if((*d_t_prev)!=NULL)
        cudaStatus = hipFree((*d_t_prev));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipFree failed!");
        return;
    }

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        getchar();
        return;
    }
}

void performCPUCFD(float *t, float *t_prev, float *cpuTime)
{

    float h,x,y;

    h = 1.0f/(Ni-1);

    for(unsigned int i=0;i<Ni;i++)
    {
        x = i*h;
        t_prev[i*Nj+0] = x*x;
        t_prev[i*Nj+(Nj-1)] = x*x + 1.0f;
    }

    for(unsigned int j=0;j < Nj; j++)
    {
        y = j*h;
        t_prev[0*Nj+j] = y*y;
        t_prev[((Ni-1) * Nj) + j] = 1.0f + y*y;

    }

    float elapsedTimeInMs = 0.0f;

    clock_t start = clock();

    for(unsigned int k=0;k<nIterations;k++)
    {
        for(unsigned int j=1;j<(Nj-1);j++)
        {
            for(unsigned int i=1;i<(Ni-1);i++)
            {
                t[i*Nj+j] = 0.25f * (t_prev[(i-1)*Nj+j] + t_prev[(i+1)*Nj+j] + t_prev[i*Nj+(j-1)] +
                                     t_prev[i*Nj+(j+1)] - 4*h*h);
            }
        }

        float* pingPong = t_prev;
        t_prev = t;
        t = pingPong;
    }

    clock_t end = clock();
    elapsedTimeInMs = (float)((end - start) * 1000 / CLOCKS_PER_SEC);

    printf("\n CPU Time:: %f ms", elapsedTimeInMs);
    *cpuTime = elapsedTimeInMs;
}

int performGPUCFD(float *d_t, float *d_t_prev, float *t, float *t_prev, float*gpuTime)
{

    float h,x,y;
    const char *str = (char*) malloc(1024); // To store error string

    //Decide how many blocks per thread and how many blocks per grid
    dim3 dimBlock(THREADS_PER_BLOCK_X,THREADS_PER_BLOCK_Y);
    dim3 dimGrid(Nj/dimBlock.x,Ni/dimBlock.y);

    h = 1.0f/(Ni-1);
    memset(t_prev, 0, sizeof(float) * Ni * Nj);

    for(unsigned int i=0;i<Ni;i++)
    {
        x = i*h;
        t_prev[i*Nj+0] = x*x;
        t_prev[i*Nj+(Nj-1)] = x*x + 1.0f;
    }

    for(unsigned int j=0;j < Nj; j++)
    {
        y = j*h;
        t_prev[0*Nj+j] = y*y;
        t_prev[((Ni-1) * Nj) + j] = 1.0f + y*y;

    }

    //Copy data to device
    hipMemcpy(d_t_prev, t_prev, sizeof(float) * Ni * Nj , hipMemcpyHostToDevice);

    //Insert event to calculate time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);


    //This calls Version 1 of kernel which uses Global memory
    if(kernelVersion ==1)
    {
        hipEventRecord(start, 0);

        for(unsigned int k=0;k<nIterations;k++)
        {
            // Launch a kernel on the GPU with one thread for each element.
            calculateCFD_V1<<<dimGrid,dimBlock>>>(d_t_prev,d_t, Ni, Nj, h);

            float* pingPong = d_t_prev;
            d_t_prev = d_t;
            d_t = pingPong;
        }
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
    }
        //This calls Version 2 of kernel which uses optimization by copying data to shared memory
    else if(kernelVersion ==2)
    {
        hipEventRecord(start, 0);

        for(unsigned int k=0;k<nIterations;k++)
        {
            // Launch a kernel on the GPU with one thread for each element.
            calculateCFD_V2<<<dimGrid,dimBlock>>>(d_t_prev,d_t, Ni, Nj, h);

            float* pingPong = d_t_prev;
            d_t_prev = d_t;
            d_t = pingPong;
        }
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);

    }

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("\n GPU Time:: %f ms", elapsedTime);

    *gpuTime = elapsedTime;

    hipError_t cudaStatus = hipMemcpy(t, d_t_prev, sizeof(float) * Ni * Nj , hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        str = hipGetErrorString(cudaStatus);
        fprintf(stderr, "CUDA Error!:: %s\n", str);
        getchar();
        return 0;
    }

    return 1;
}

int checkHostEqualsDevice(float* o_host, float* o_device)
{
    int flag =1;

    float tolerance = 0.0001f;
    //Compare the results
    for(unsigned int j=0;j<Nj;j++)
    {
        for(unsigned int i=0;i<Ni;i++)
        {
            if( (o_host[i*Nj+j] - o_device[i*Nj+j]) >= tolerance ||  (o_host[i*Nj+j] - o_device[i*Nj+j]) <= -tolerance)
            {
                printf("\n D=[%f]!=H=[%f] since Diff  > tol %f for [%d][%d]",o_device[i*Nj+j],  o_host[i*Nj+j],tolerance, i, j);
                flag =0;
                //getchar();
            }
        }
    }

    return flag;
}