#include "hip/hip_runtime.h"
#include "definitions.cuh"
#include "stdio.h"
#define RADIUS 1
//Performs CFD calculation on global memory. This code does not use any advance optimization technique on GPU
// But still acheives many fold performance gain
__global__ void calculateCFD_V1( float* input,  float* output, unsigned int Ni, unsigned int Nj,
								 float h)
{
	unsigned int i = blockDim.x * blockIdx.x + threadIdx.x; // Y - ID
	unsigned int j = blockDim.y * blockIdx.y + threadIdx.y; // X - ID

	unsigned int iPrev = i-1; // Previous Y element
	unsigned int iNext = i+1; // Next Y element

	unsigned int jPrev = j-1; //Previous X element
	unsigned int jNext = j+1; // Next X element


	unsigned int index = i * Nj + j;

	if( i > 0 && j > 0 && i < (Ni-1) && j <(Nj-1))
		output[index] = 0.25f * (input[iPrev * Nj + j] + input[iNext* Nj + j] + input[i * Nj+ jPrev]
								 + input[i* Nj + jNext] - 4*h*h);
}

//This version of Kernel uses optimization by copying the data into shared memory and hence results in better performance
__global__ void calculateCFD_V2( float* input,  float* output, unsigned int Ni, unsigned int Nj, 
								   float h){

	//printf("\nthread_per_block:x:%d,y:%d\n",blockDim.x,blockDim.y);
	//32,16
	//Current Global ID
	unsigned int i = blockDim.x * blockIdx.x + threadIdx.x; // Y - ID
	unsigned int j = blockDim.y * blockIdx.y + threadIdx.y; // X - ID

	unsigned int xlindex = threadIdx.x+RADIUS;
	unsigned int ylindex = threadIdx.y+RADIUS;
	// Fill the size of shared memory
	__shared__ float sData [2*RADIUS+THREADS_PER_BLOCK_X][2*RADIUS+THREADS_PER_BLOCK_Y];

    unsigned int index = (i)* Nj + (j) ;
    sData[xlindex][ylindex] = input[index];

	if (threadIdx.x < RADIUS) {
		if(blockIdx.x > 0)
			sData[xlindex - RADIUS][ylindex] = input[index-Ni*RADIUS];
		if(blockIdx.x < (gridDim.x-1))
			sData[xlindex + THREADS_PER_BLOCK_X][ylindex] = input[index + THREADS_PER_BLOCK_X*Ni];
	}
	if (threadIdx.y < RADIUS)
	{
        if(blockIdx.y > 0)
	        sData[xlindex][ylindex - RADIUS] = input[index - RADIUS];
        if(blockIdx.y < (gridDim.y - 1))
        sData[xlindex][ylindex + THREADS_PER_BLOCK_Y] = input[index + THREADS_PER_BLOCK_Y];
	}



	__syncthreads();
	//Add synchronization. Guess Why?

	if( i > 0 && j > 0 && i < (Ni-1) && j <(Nj-1))
		output[index] = 0.25f * (sData[xlindex-1][ylindex] + sData[xlindex+1][ylindex] + sData[xlindex][ylindex-1]
			+ sData[xlindex][ylindex+1] - 4*h*h);

}

/*
 __global__ void calculateCFD_V2( float* input,  float* output, unsigned int Ni, unsigned int Nj,
								   float h){

	//printf("\nthread_per_block:x:%d,y:%d\n",blockDim.x,blockDim.y);
	//32,16
	//Current Global ID
	unsigned int j = blockDim.x * blockIdx.x + threadIdx.x; // Y - ID
	unsigned int i = blockDim.y * blockIdx.y + threadIdx.y; // X - ID

	unsigned int xlindex = threadIdx.x+RADIUS;
	unsigned int ylindex = threadIdx.y+RADIUS;
	// Fill the size of shared memory
	__shared__ float sData [2*RADIUS+THREADS_PER_BLOCK_Y][2*RADIUS+THREADS_PER_BLOCK_X];

    unsigned int index = (i)* Nj + (j) ;
    sData[ylindex][xlindex] = input[index];

	if (threadIdx.x < RADIUS) {
		if(blockIdx.x > 0)
			sData[ylindex][xlindex - RADIUS] = input[index-RADIUS];
		if(blockIdx.x < (gridDim.x-1))
			sData[ylindex][xlindex + THREADS_PER_BLOCK_X] = input[index + THREADS_PER_BLOCK_X];
	}
	if (threadIdx.y < RADIUS)
	{
        if(blockIdx.y > 0)
	        sData[ylindex - RADIUS][xlindex] = input[index - RADIUS*Ni];
        if(blockIdx.y < (gridDim.y - 1))
            sData[ylindex + THREADS_PER_BLOCK_Y][xlindex] = input[index + THREADS_PER_BLOCK_Y*Ni];
	}



	__syncthreads();
	//Add synchronization. Guess Why?

	if( i > 0 && j > 0 && i < (Ni-1) && j <(Nj-1))
		output[index] = 0.25f * (sData[ylindex-1][xlindex] + sData[ylindex+1][xlindex] + sData[ylindex][xlindex-1]
			+ sData[ylindex][xlindex+1] - 4*h*h);

}
 */