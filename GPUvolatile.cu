
#include <hip/hip_runtime.h>
#include <vector>
#include <algorithm>
#include <cstdio>
#include <iostream>
#include <chrono>
#include <fstream>
#include <nvml.h>
#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>
const long long tdelay=1000000LL;
#define SIZE  1
__global__ void dkern(){

    long long start = clock64();
    while(clock64() < start+tdelay);
}
using namespace std;

int main() {

    for (int i=0;i<100;i++){
        void *deviceArray=(void*)0;
        hipMalloc((void**)&deviceArray,1e9);
    }

    {
        int my_delay = 100;
        for (int i = 0; ; i++){
            dkern<<<1,1>>>();
            usleep(my_delay);
        }
    }





    return 0;
}