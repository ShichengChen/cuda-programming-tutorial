
#include <hip/hip_runtime.h>
#include <vector>
#include <algorithm>
#include <cstdio>
#include <iostream>
#include <chrono>
#include <fstream>
#include <nvml.h>
#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>
const long long tdelay=1000000LL;
#define SIZE  1
__global__ void dkern(){

    long long start = clock64();
    while(clock64() < start+tdelay);
}
using namespace std;

int main() {
    //freopen ("swapinout.txt","w",stdout);
    {
        int my_delay = 100;
        for (int i = 0;i<100 ; i++){
            dkern<<<1,1>>>();
            usleep(my_delay);
        }
    }





    return 0;
}