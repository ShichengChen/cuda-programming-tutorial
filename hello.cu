
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ const char *STR = "HELLO WORLD!";
const char STR_LENGTH = 12;

__global__ void hello()
{
	
	printf("calling kernel\n");
	printf("%c\n", STR[threadIdx.x % STR_LENGTH]);
	printf("kernel called\n");
}

int main(void)
{
	int num_threads = STR_LENGTH;
	int num_blocks = 1;
	printf("before hello\n");
	hello<<<num_blocks,num_threads>>>();
	printf("after hello\n");
	hipDeviceSynchronize();


{
    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr != hipSuccess)
        printf("kernel launch failed with error \"%s\".\n",
               hipGetErrorString(cudaerr));
}

	return 0;
}
