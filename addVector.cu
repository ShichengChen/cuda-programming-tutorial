
#include <hip/hip_runtime.h>
#include <stdio.h>

/* experiment with N */
/* how large can it be? */
#define N (2048*2048)
#define THREADS_PER_BLOCK 512


__global__ void vector_add(int *a, int *b, int *c)
{
    /* insert code to calculate the index properly using blockIdx.x, blockDim.x, threadIdx.x */
    int index = int(blockIdx.x) * int(blockDim.x) + int(threadIdx.x);
    //printf("block dim:%d,%d,blockid:%d,threadidx.x:%d\n",blockDim.x,blockDim.y,blockIdx.x,threadIdx.x);
    //printf("index:%d\n",index);
    c[index] = a[index] + b[index];
    if(index >= N - 1)
    {
        printf("%d\n",index);
        printf("%d,%d,%d\n",a[index],b[index],c[index]);
    }
}


int main()
{
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;

    int size = N * sizeof( int );

    /* allocate space for device copies of a, b, c */
    hipMalloc( (void **) &d_a, size );
    hipMalloc( (void **) &d_b, size );
    hipMalloc( (void **) &d_c, size );

    /* allocate space for host copies of a, b, c and setup input values */

    a = (int *)malloc( size );
    b = (int *)malloc( size );
    c = (int *)malloc( size );

    for( int i = 0; i < N; i++ )
    {
        a[i] = b[i] = i;
        c[i] = 0;
    }

    /* copy inputs to device */
    /* fix the parameters needed to copy data to the device */
    hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
    hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);

    /* launch the kernel on the GPU */
    /* insert the launch parameters to launch the kernel properly using blocks and threads */
    vector_add<<< N/THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>( d_a, d_b, d_c );

    /* copy result back to host */
    /* fix the parameters needed to copy data back to the host */
    hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);


    printf( "c[0] = %d\n",c[0] );
    printf( "c[%d] = %d\n",N-1, c[N-1] );

    /* clean up */
    {
        hipError_t cudaerr = hipDeviceSynchronize();
        if (cudaerr != hipSuccess)
            printf("kernel launch failed with error \"%s\".\n",
                   hipGetErrorString(cudaerr));
    }

    free(a);
    free(b);
    free(c);
    hipFree( d_a );
    hipFree( d_b );
    hipFree( d_c );

    return 0;
} /* end main */