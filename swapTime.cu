
#include <hip/hip_runtime.h>
#include <vector>
#include <algorithm>
#include <cstdio>
#include <iostream>
#include <chrono>
#include <fstream>
#define SIZE  1
using namespace std;

int main() {
    int *a,*b,*c;
    long long a0 = std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
    hipMallocManaged(&a, SIZE * sizeof(int));
    long long b0 = std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
    hipMallocManaged(&b, SIZE * sizeof(int));
    long long c0 = std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
    hipMallocManaged(&c, SIZE * sizeof(int));
    long long d0 = std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
    cout<<"d-c:" << d0-c0<< endl;
    hipFree(a);
    hipFree(b);
    hipFree(c);
    return 0;
}