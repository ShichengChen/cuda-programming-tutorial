
#include <hip/hip_runtime.h>
#include <vector>
#include <algorithm>
#include <cstdio>
#include <iostream>
#include <chrono>
#include <fstream>
#define SIZE  1
using namespace std;

int main() {
    freopen ("swapinout.txt","w",stdout);
    int s[] = {1228800 ,26214400, 26214400, 26214400, 26214400, 26214400 ,26214400  ,6553600,
               13107200, 13107200, 13107200, 13107200 ,13107200 ,13107200 , 3276800 , 1179648,
               6553600 , 6553600 , 6553600 , 2359296 , 6553600 , 6553600 , 6553600 , 2359296,
               6553600 , 6553600 , 6553600 , 1638400 , 4718592 , 3276800 , 3276800 , 3276800,
               9437184 , 3276800 , 3276800 , 1048576 , 1048576 , 9437184 , 9437184 , 9437184,
               9437184 , 9437184 , 9437184 , 9437184 , 3276800 , 9437184 , 3276800};
    cout << "size,in,out" << endl;
    for (int i = 0;i < 47;i++){
        long long size = s[i];
        void *hostArray=(void*)0;
        hipHostMalloc(&hostArray,size, hipHostMallocDefault);
        void *deviceArray=(void*)0;
        hipMalloc((void**)&deviceArray,size);


        long long a0 = std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
        hipMemcpy(deviceArray,hostArray,size,hipMemcpyHostToDevice);
        long long b0 = std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
        hipMemcpy(hostArray,deviceArray,size,hipMemcpyDeviceToHost);
        long long c0 = std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
        cout <<s[i]<< ","<<b0-a0 << "," << c0-b0 << endl;

    }



    return 0;
}