#include "hip/hip_runtime.h"
#include <vector>
#include <algorithm>
#include <cstdio>
#include <iostream>
#include <chrono>
#include <fstream>
#include <nvml.h>
#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>
const long long tdelay=1000000LL;
#define SIZE  1
__global__ void dkern(){

    long long start = clock64();
    while(clock64() < start+tdelay);
}
using namespace std;

int main() {
    //freopen ("swapinout.txt","w",stdout);
//    {
//        int my_delay = 100;
//        for (int i = 0; ; i++){
//            dkern<<<1,1>>>();
//            usleep(my_delay);
//        }
//    }





    int nDevices;

    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (KHz): %d\n",
               prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n",
               prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
               2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
        cout << " deviceOverlap: " << prop.deviceOverlap << endl;
        cout << " asyncEngineCount: " << prop.asyncEngineCount << endl;

    }


    while(true)
    {
        int s[] = {1,4718592,9437184};
        cout << "size,in,out" << endl;
        for (int i = 0;i < 2;i++){
            long long size = s[i];
            void *hostArray=(void*)0;
            hipHostMalloc(&hostArray,size);
            void *deviceArray=(void*)0;
            hipMalloc((void**)&deviceArray,size);


            long long a0 = std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
            hipMemcpy(deviceArray,hostArray,size,hipMemcpyHostToDevice);
            long long b0 = std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
            hipMemcpy(hostArray,deviceArray,size,hipMemcpyDeviceToHost);
            long long c0 = std::chrono::duration_cast<std::chrono::nanoseconds>(std::chrono::system_clock::now().time_since_epoch()).count();
            cout <<s[i]<< ","<<b0-a0 << "," << c0-b0 << endl;

        }
        break;
    }




    return 0;
}